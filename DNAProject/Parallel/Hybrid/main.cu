#include "hip/hip_runtime.h"
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/transform_reduce.h>
#include <thrust/functional.h>
#include <thrust/transform.h>
#include <thrust/copy.h>
#include <thrust/fill.h>
#include <thrust/sequence.h>
#include <thrust/execution_policy.h>
#include <iostream>
#include <random>
#include <vector>
#include <string>
#include <algorithm>
#include <chrono>
#include <cstdio>
#include <omp.h>

#define MATCH 2
#define MISMATCH -1
#define GAP -1
#define SHOW_REPORT false
#define SHOW_MATRIX false

struct score {
    // Character of row;
    char y;
    // Constructor to store char internally
    score(char y) : y(y) { }
    __host__ __device__
    int operator()(const thrust::tuple<char, int, int> t){
        char x = t.get<0>();
        int H_up = t.get<1>();
        int H_diag = t.get<2>();
        // printf("Strings: %s;%s", a, b);
        int up, diag;
        // Branchless evaluation of three possibilities
        up = (H_up+GAP); // Above
        diag = (x != y)*(H_diag+MISMATCH) + (x == y)*(H_diag+MATCH); // Diag
        // printf("Eval: up=(%d); diag=(%c,%c,%d); gap=%d; match=%d; miss=%d;\n", H_up, x, y, H_diag, gap, match, miss);
        // Branchless evaluation of max
        int max = (up > diag)*up +
                  (diag >= up)*diag;
        max = (max>0)*max;
        return max;
    }
};

template <typename T>
struct PrintVec {
    __host__ __device__
    T operator()(const T y){
        // Unpack to Sij, Si-1j
        printf("%d ", y);
        return y;
    }
};

struct line_update {
    __host__ __device__
    int operator()(const int& x, const int& y){
        // Unpack to Sij, Si-1j
        int curr = y;
        int left = x-1;
        // Hand compute max
        int max = (left >= curr)*left + (curr > left)*curr;
        max = (max>0)*max;
        // if(max != 0) printf("X = %d Y = %d LEFT = %d CURR = %d OUT = %d\n", x, y, left, curr, max);
        return max;
    }
};

int smithwaterman_score_seq(
    thrust::detail::normal_iterator<thrust::device_ptr<int>> UpperRow_begin,
    thrust::detail::normal_iterator<thrust::device_ptr<int>> UpperRow_end,
    thrust::detail::normal_iterator<thrust::device_ptr<int>> Temp_begin,
    thrust::detail::normal_iterator<thrust::device_ptr<int>> Temp_end,
    thrust::detail::normal_iterator<thrust::device_ptr<char>> bigstr_begin,
    thrust::detail::normal_iterator<thrust::device_ptr<char>> bigstr_end,
    thrust::device_vector<char> smallstr,
    int smallStrStart, int smallStrEnd
) {
    thrust::fill(UpperRow_begin, UpperRow_end, 0);
    
    int high_score = 0;
    int curr_score = 0;
    // Debug printing
    if (SHOW_MATRIX){
        thrust::transform(UpperRow_begin, UpperRow_end, UpperRow_begin, PrintVec<int>());
        std::cout << std::endl;
    }
    for(int i = smallStrStart; i<smallStrEnd; i++){
        // First stage, check against above row
        thrust::transform(
            thrust::make_zip_iterator(thrust::make_tuple(bigstr_begin+1, UpperRow_begin+1, UpperRow_begin)),
            thrust::make_zip_iterator(thrust::make_tuple(bigstr_end, UpperRow_end, UpperRow_end-1)),
            Temp_begin+1, 
            score(smallstr[i])
        );

        // Debug printing
        // std::cout << std::endl;
        // thrust::transform(Temp_begin, Temp_end, Temp_begin, PrintVec<int>());
        // std::cout << std::endl;

        // Second stage, check against self left 
        thrust::inclusive_scan(
            Temp_begin,
            Temp_end,
            UpperRow_begin, 
            line_update()
        );
        
        // Debug printing
        if (SHOW_MATRIX){
            thrust::transform(UpperRow_begin, UpperRow_end, UpperRow_begin, PrintVec<int>());
            std::cout << std::endl;
        }

        // Keep track of best score
        curr_score = thrust::reduce(UpperRow_begin, UpperRow_end, 0, thrust::maximum<int>());
        if (curr_score > high_score) high_score = curr_score;
    }
    return high_score;
    // return 0;
}

int main(){

    // Inputs
    int len_a, len_b;
    std::cin >> len_a >> len_b;
    std::string a, b;
    std::cin >> a >> b;
    std::string big, small;
    a = "-" + a;
    b = "-" + b;

    if(len_a > len_b){
        big = a;
        small = b;
    } else {
        big = b;
        small = a;
    }

    auto begin_Parallel = std::chrono::high_resolution_clock::now();
    // Parallel

    // Send string to gpu vector    
    thrust::device_vector<char> bigstr(big.length());
    for(int i = 0; i < big.length(); i++){
        bigstr[i] = big[i];
    }

    thrust::device_vector<char> smallstr(small.length());
    for(int i = 0; i < small.length(); i++){
        smallstr[i] = small[i];
    }

    thrust::device_vector<int> UpperRow(big.length());
    thrust::device_vector<int> Temp(big.length());
    
    int curr_size;
    int smallStringStart, smallStringEnd, bigStringStart, bigStringEnd;
    int high_score = MISMATCH * len_a * len_b; 
    int score = 0;
    #pragma omp parallel for reduction(max:high_score)
    for(curr_size = small.length()-1; curr_size>0; curr_size--){
        for (int i = 1; i<small.length()-curr_size+1; i++){
            for (int j = 1; j<big.length()-curr_size+1; j++){
                smallStringStart = i;
                smallStringEnd = i + curr_size;
                bigStringStart = j;
                bigStringEnd = j + curr_size;
                score = smithwaterman_score_seq(
                    UpperRow.begin(), UpperRow.begin()+curr_size, 
                    Temp.begin(), Temp.begin()+curr_size,
                    bigstr.begin() + bigStringStart, bigstr.begin() + bigStringEnd,
                    smallstr, smallStringStart, smallStringEnd
                );
                high_score = score > high_score ? score : high_score;
            }
        }
        curr_size = 0;
        curr_size --;
    }

    // Output
    auto end_Parallel = std::chrono::high_resolution_clock::now();
    auto elapsed_Parallel = std::chrono::duration_cast<std::chrono::nanoseconds>(end_Parallel - begin_Parallel);
    if (SHOW_REPORT){
        std::cout << "Big string: " << big << std::endl;
        std::cout << "Small string: " << small << std::endl;

        std::cout << "=======================================" << std::endl;
        std::cout << "Parallel: " << std::endl;
        std::cout << "=======================================" << std::endl;
        printf("Time measured: %.6f seconds.\n", elapsed_Parallel.count() * 1e-9);
        std::cout << "Score: " << high_score << std::endl;
    } else {
        std::cout << high_score << std::endl;
    }
    return 0;
}