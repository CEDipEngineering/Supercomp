#include "hip/hip_runtime.h"
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/transform_reduce.h>
#include <thrust/functional.h>
#include <thrust/transform.h>
#include <thrust/copy.h>
#include <thrust/fill.h>
#include <thrust/sequence.h>
#include <thrust/execution_policy.h>
#include <iostream>
#include <random>
#include <vector>
#include <string>
#include <algorithm>
#include <chrono>
#include <cstdio>

#define MATCH 2
#define MISMATCH -1
#define GAP -1
#define SHOW_REPORT false
#define SHOW_MATRIX true

struct score {
    // Character of row;
    char y;
    // Constructor to store char internally
    score(char y) : y(y) { }
    __host__ __device__
    int operator()(const thrust::tuple<char, int, int> t){
        char x = t.get<0>();
        int H_up = t.get<1>();
        int H_diag = t.get<2>();
        // printf("Strings: %s;%s", a, b);
        int up, diag;
        // Branchless evaluation of three possibilities
        up = (H_up+GAP); // Above
        diag = (x != y)*(H_diag+MISMATCH) + (x == y)*(H_diag+MATCH); // Diag
        // printf("Eval: up=(%d); diag=(%c,%c,%d); gap=%d; match=%d; miss=%d;\n", H_up, x, y, H_diag, gap, match, miss);
        // Branchless evaluation of max
        int max = (up > diag)*up +
                  (diag >= up)*diag;
        max = (max>0)*max;
        return max;
    }
};

struct PrintVec {
    __host__ __device__
    int operator()(const int y){
        // Unpack to Sij, Si-1j
        printf("%d ", y);
        return y;
    }
};

struct line_update {
    __host__ __device__
    int operator()(const int& x, const int& y){
        // Unpack to Sij, Si-1j
        int curr = y;
        int left = x-1;
        // Hand compute max
        int max = (left >= curr)*left + (curr > left)*curr;
        max = (max>0)*max;
        // if(max != 0) printf("X = %d Y = %d LEFT = %d CURR = %d OUT = %d\n", x, y, left, curr, max);
        return max;
    }
};


int main(){

    // Inputs
    int len_a, len_b;
    std::cin >> len_a >> len_b;
    std::string a, b;
    std::cin >> a >> b;
    std::string big, small;
    a = "-" + a;
    b = "-" + b;

    if(len_a > len_b){
        big = a;
        small = b;
    } else {
        big = b;
        small = a;
    }

    auto begin_Parallel = std::chrono::high_resolution_clock::now();
    // Parallel

    // Create and fill both row vectors with zero
    thrust::device_vector<int> UpperRow(big.length());
    thrust::device_vector<int> Temp(big.length());
    thrust::fill(UpperRow.begin(), UpperRow.end(), 0);
    thrust::fill(Temp.begin(), Temp.end(), 0);

    // Send string to gpu vector    
    thrust::device_vector<char> big_str(big.length());
    for(int i = 0; i < big.length(); i++){
        big_str[i] = big[i];
    }

    int high_score = 0;
    int curr_score = 0;
    // Debug printing
    if (SHOW_MATRIX){
        thrust::transform(UpperRow.begin(), UpperRow.end(), UpperRow.begin(), PrintVec());
        std::cout << std::endl;
    }
    for(int i = 0; i<small.length()-1; i++){
        
        // First stage, check against above row
        thrust::transform(
            thrust::make_zip_iterator(thrust::make_tuple(big_str.begin()+1, UpperRow.begin()+1, UpperRow.begin())),
            thrust::make_zip_iterator(thrust::make_tuple(big_str.end(), UpperRow.end(), UpperRow.end()-1)),
            Temp.begin()+1, 
            score(small[i+1])
        );

        // Debug printing
        // std::cout << std::endl;
        // thrust::transform(Temp.begin(), Temp.end(), Temp.begin(), PrintVec());
        // std::cout << std::endl;

        // Second stage, check against self left 
        thrust::inclusive_scan(
            Temp.begin(),
            Temp.end(),
            UpperRow.begin(), 
            line_update()
        );
        
        // Debug printing
        if (SHOW_MATRIX){
            thrust::transform(UpperRow.begin(), UpperRow.end(), UpperRow.begin(), PrintVec());
            std::cout << std::endl;
        }

        // Keep track of best score
        curr_score = thrust::reduce(UpperRow.begin(), UpperRow.end(), 0, thrust::maximum<int>());
        if (curr_score > high_score) high_score = curr_score;
    }

    // Output
    auto end_Parallel = std::chrono::high_resolution_clock::now();
    auto elapsed_Parallel = std::chrono::duration_cast<std::chrono::nanoseconds>(end_Parallel - begin_Parallel);
    if (SHOW_REPORT){
        std::cout << "Big string: " << big << std::endl;
        std::cout << "Small string: " << small << std::endl;

        std::cout << "=======================================" << std::endl;
        std::cout << "Parallel: " << std::endl;
        std::cout << "=======================================" << std::endl;
        printf("Time measured: %.6f seconds.\n", elapsed_Parallel.count() * 1e-9);
        std::cout << "Score: " << high_score << std::endl;
    } else {
        std::cout << high_score << std::endl;
    }

    return 0;
}