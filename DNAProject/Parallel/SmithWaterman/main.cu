#include "hip/hip_runtime.h"
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/transform_reduce.h>
#include <thrust/functional.h>
#include <thrust/transform.h>
#include <thrust/copy.h>
#include <thrust/fill.h>
#include <thrust/sequence.h>
#include <thrust/execution_policy.h>
#include <iostream>
#include <random>
#include <vector>
#include <string>
#include <algorithm>
#include <omp.h>
#include <chrono>
#include <cstdio>
#define MATCH 2
#define MISMATCH -1
#define GAP -1
#define SHOW_REPORT false

struct line_process { 
    // Access to both strings
    const char *a, *b;
    // Constructor to store them internally
    line_process(const char* a, const char* b) : a(a), b(b) { }
    __host__ __device__
    int operator()(const thrust::tuple<int, int, int>& y){
        int diag = y.get<0>();
        int up = y.get<1>();
        int i = y.get<2>();
        int j = y.get<2>();
        printf("i: %d; j: %d; up: %d; diag: %d;\n", i,j,up,diag);
        int gap, match, miss;
        // Branchless evaluation of three possibilities
        gap = (a[i] == '-' || b[j-1] == '-') * (up - 1);
        match = (a[i-1] == b[j-1]) * (diag + 2);
        miss = (a[i-1] != b[j-1]) * (diag - 1);
        // std::max not allowed, hand compute
        int max = gap;
        if (match > max) max = match;
        if (miss > max) max = miss;
        if (max < 0) max = 0;
        return max; // Can never be negative, since at least 1 of the above will be 0
    }
};

struct score {
    __host__ __device__
    int operator()(const thrust::tuple<char, char, int> t){
        char x = t.get<0>();
        char y = t.get<1>();
        int H = t.get<2>();
        // printf("Strings: %s;%s", a, b);
        int gap, match, miss;
        // Branchless evaluation of three possibilities
        gap = (x == '-' || y == '-');
        match = (x == y && !gap);
        miss = (x != y && !gap);
        printf("Eval: chars=(%c,%c,%d); gap=%d; match=%d; miss=%d;\n", x, y, H, gap, match, miss);
        int max = gap;
        if (match > max) max = match;
        if (miss > max) max = miss;
        if (max < 0) max = 0;
        return max;
    }
};

struct line_update {
    __host__ __device__
    int operator()(const thrust::tuple<int, int>& y){
        // Unpack to Sij, Si-1j
        int curr = y.get<0>();
        int left = y.get<1>()-1;
        // Hand compute max
        int max = curr;
        if (left > max) max = left;
        if (max < 0) max = 0;
        return max;
    }
};


int main(){

    // Inputs
    int len_a, len_b;
    std::cin >> len_a >> len_b;
    std::string a, b;
    std::cin >> a >> b;
    std::string big, small;
    a = "-" + a;
    b = "-" + b;

    if(len_a > len_b){
        big = a;
        small = b;
    } else {
        big = b;
        small = a;
    }

    auto begin_Parallel = std::chrono::high_resolution_clock::now();
    // Parallel

    std::cout << "Big string: " << big << std::endl;
    std::cout << "Small string: " << small << std::endl;

    thrust::device_vector<int> UpperRow(big.length());
    thrust::device_vector<int> Temp(big.length());

    thrust::fill(UpperRow.begin(), UpperRow.end(), 0);
    thrust::fill(Temp.begin(), Temp.end(), 0);

    // thrust::counting_iterator<int> first(0);
    // thrust::counting_iterator<int> last = first + big.length();
    
    // thrust::transform(
    //     thrust::make_zip_iterator(thrust::make_tuple(UpperRow.begin(), UpperRow.begin()+1, first)), 
    //     thrust::make_zip_iterator(thrust::make_tuple(UpperRow.end()-1, UpperRow.end(), last)), 
    //     Temp.begin()+1, 
    //     line_process((big.c_str()), (small.c_str()))
    // );
    
    thrust::device_vector<char> big_str(big.length());
    for(int i = 0; i < big.length(); i++){
        big_str[i] = big[i];
    }
    thrust::device_vector<char> small_str(small.length());
    for(int i = 0; i < small.length(); i++){
        small_str[i] = small[i];
    }
    // thrust::generate(big_str.begin(), big_str.end(), thrust::make_constant_iterator<char>(22));
    
    
    thrust::transform(
        thrust::make_zip_iterator(thrust::make_tuple(big_str.begin(), small_str.begin(), UpperRow.begin())),
        thrust::make_zip_iterator(thrust::make_tuple(big_str.end(), small_str.end(), UpperRow.end())),
        big_str.begin(), 
        score()
    );
    std::cout << std::endl;


    // thrust::transform(thrust::make_counting_iterator(0), thrust::make_counting_iterator(3), X.begin(), score(const_cast<char *>(big.c_str()), const_cast<char *>(small.c_str())));


    // Output
    auto end_Parallel = std::chrono::high_resolution_clock::now();
    auto elapsed_Parallel = std::chrono::duration_cast<std::chrono::nanoseconds>(end_Parallel - begin_Parallel);
    // if (SHOW_REPORT){
    //     std::cout << "Big string: " << big << std::endl;
    //     std::cout << "Small string: " << small << std::endl;

    //     std::cout << "=======================================" << std::endl;
    //     std::cout << "Parallel: " << std::endl;
    //     std::cout << "=======================================" << std::endl;
    //     printf("Time measured: %.6f seconds.\n", elapsed_Parallel.count() * 1e-9);
    //     std::cout << "Match string 1: " << best_a_vec_par[best_ix_par] << std::endl;
    //     std::cout << "Match string 2: " << best_b_vec_par[best_ix_par] << std::endl;
    //     std::cout << "Score: " << scores_par[best_ix_par] << std::endl;
    // } else {
    //     std::cout << scores_par[best_ix_par] << std::endl;
    // }

    return 0;
}