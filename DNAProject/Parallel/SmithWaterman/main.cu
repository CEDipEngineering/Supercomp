#include "hip/hip_runtime.h"
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/transform_reduce.h>
#include <thrust/functional.h>
#include <thrust/transform.h>
#include <thrust/copy.h>
#include <thrust/fill.h>
#include <thrust/sequence.h>
#include <thrust/execution_policy.h>
#include <iostream>
#include <random>
#include <vector>
#include <string>
#include <algorithm>
#include <omp.h>
#include <chrono>
#include <cstdio>
#define MATCH 2
#define MISMATCH -1
#define GAP -1
#define SHOW_REPORT false

struct line_process { 
    // Access to both strings
    const char *a, *b;
    // Constructor to store them internally
    line_process(const char* a, const char* b) : a(a), b(b) { }
    __host__ __device__
    int operator()(const thrust::tuple<int, int, int>& y){
        int diag = y.get<0>();
        int up = y.get<1>();
        int i = y.get<2>();
        int j = y.get<2>();
        printf("i: %d; j: %d; up: %d; diag: %d;\n", i,j,up,diag);
        int gap, match, miss;
        // Branchless evaluation of three possibilities
        gap = (a[i] == '-' || b[j-1] == '-') * (up - 1);
        match = (a[i-1] == b[j-1]) * (diag + 2);
        miss = (a[i-1] != b[j-1]) * (diag - 1);
        // std::max not allowed, hand compute
        int max = gap;
        if (match > max) max = match;
        if (miss > max) max = miss;
        if (max < 0) max = 0;
        return max; // Can never be negative, since at least 1 of the above will be 0
    }
};

struct score {

    char y;
    // Constructor to store them internally
    score(char y) : y(y) { }
    __host__ __device__
    int operator()(const thrust::tuple<char, int, int> t){
        char x = t.get<0>();
        int H_up = t.get<1>();
        int H_diag = t.get<2>();
        // printf("Strings: %s;%s", a, b);
        int gap, match, miss;
        // Branchless evaluation of three possibilities
        gap = (y == '-')*(H_up+GAP);
        match = (x == y && !(y == '-'))*(H_diag+MATCH);
        miss = (x != y && !(y == '-'))*(H_diag+MISMATCH);
        printf("Eval: up=(%d); diag=(%c,%c,%d); gap=%d; match=%d; miss=%d;\n", H_up, x, y, H_diag, gap, match, miss);
        // Branchless evaluation of max
        int max = ((gap >= match) && (gap > miss))*gap +
                  ((match > gap) && (match >= miss))*match +
                  ((miss >= gap) && (miss > match))*miss;
        max = (max>0)*max;
        return max;
    }
};

struct PrintVec {
    __host__ __device__
    int operator()(const int y){
        // Unpack to Sij, Si-1j
        printf("%d ", y);
        return y;
    }
};

struct line_update {
    __host__ __device__
    int operator()(const int& x, const int& y){
        // Unpack to Sij, Si-1j
        int curr = y;
        int left = x-1;
        // Hand compute max
        int max = (left > curr)*left + (curr > left)*curr;
        max = (max>0)*max;
        return max;
    }
};


int main(){

    // Inputs
    int len_a, len_b;
    std::cin >> len_a >> len_b;
    std::string a, b;
    std::cin >> a >> b;
    std::string big, small;
    a = "-" + a;
    b = "-" + b;

    if(len_a > len_b){
        big = a;
        small = b;
    } else {
        big = b;
        small = a;
    }

    auto begin_Parallel = std::chrono::high_resolution_clock::now();
    // Parallel

    // std::cout << "Big string: \t\t" << big << std::endl;
    // std::cout << "Small string: \t\t" << small << std::endl;

    thrust::device_vector<int> UpperRow(big.length());
    thrust::device_vector<int> Temp(big.length());

    thrust::fill(UpperRow.begin(), UpperRow.end(), 0);
    thrust::fill(Temp.begin(), Temp.end(), 0);
    
    thrust::device_vector<char> big_str(big.length());
    for(int i = 0; i < big.length(); i++){
        big_str[i] = big[i];
    }
    // thrust::device_vector<char> small_str(small.length());
    // for(int i = 0; i < small.length(); i++){
    //     small_str[i] = small[i];
    // }

    int high_score = 0;
    int curr_score = 0;
    for(int i = 0; i<small.length()-1; i++){
        // First stage, check against above row
        thrust::transform(
            thrust::make_zip_iterator(thrust::make_tuple(big_str.begin()+1, UpperRow.begin()+1, UpperRow.begin())),
            thrust::make_zip_iterator(thrust::make_tuple(big_str.end(), UpperRow.end(), UpperRow.end()-1)),
            Temp.begin()+1, 
            score(small[i+1])
        );
        std::cout << std::endl;
        // thrust::transform(Temp.begin(), Temp.end(), Temp.begin(), PrintVec());
        // std::cout << std::endl;
        // Second stage, check against self left 
        thrust::inclusive_scan(
            Temp.begin(),
            Temp.end(),
            UpperRow.begin(), 
            line_update()
        );
        // thrust::transform(UpperRow.begin(), UpperRow.end(), UpperRow.begin(), PrintVec());
        // std::cout << std::endl;
        curr_score = thrust::reduce(UpperRow.begin(), UpperRow.end(), 0, thrust::maximum<int>());
        if (curr_score > high_score) high_score = curr_score;
    }
    std::cout << "Highest score was: " << high_score << std::endl;


   // Output
    auto end_Parallel = std::chrono::high_resolution_clock::now();
    auto elapsed_Parallel = std::chrono::duration_cast<std::chrono::nanoseconds>(end_Parallel - begin_Parallel);
    // if (SHOW_REPORT){
    //     std::cout << "Big string: " << big << std::endl;
    //     std::cout << "Small string: " << small << std::endl;

    //     std::cout << "=======================================" << std::endl;
    //     std::cout << "Parallel: " << std::endl;
    //     std::cout << "=======================================" << std::endl;
    //     printf("Time measured: %.6f seconds.\n", elapsed_Parallel.count() * 1e-9);
    //     std::cout << "Match string 1: " << best_a_vec_par[best_ix_par] << std::endl;
    //     std::cout << "Match string 2: " << best_b_vec_par[best_ix_par] << std::endl;
    //     std::cout << "Score: " << scores_par[best_ix_par] << std::endl;
    // } else {
    //     std::cout << scores_par[best_ix_par] << std::endl;
    // }

    return 0;
}