#include "hip/hip_runtime.h"
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/transform_reduce.h>
#include <thrust/functional.h>
#include <thrust/transform.h>
#include <iostream>
#include <fstream>
#include <iomanip>
#include <cstdlib>
#include <chrono>
#include <cstdlib>
#include <algorithm>
#include <cmath>
#include <random>

using namespace std::chrono;

void reportTime(const char* msg, steady_clock::duration span) {
    auto ms = duration_cast<milliseconds>(span);
    std::cout << msg << " - levou - " <<
    ms.count() << " milisegundos" << std::endl;
}

// CRIE UMA FUNCTOR PARA CALCULAR A SQUARE
struct square
{
    __host__ __device__
    double operator()(const double& x) {
        return x * x;
    }
};

template<typename T>
struct unary_square
{
  __host__ __device__ 
  T operator()(const T &x) const
  {
    return x * x;
  }
};



// IMPLEMENTE O CALCULO DA MAGNITUDE COM THRUST
double magnitude(thrust::device_vector<double> x) {
    double result;
    thrust::device_vector<double> x2(x.size()); // Output
    thrust::transform(x.begin(), x.end(), x2.begin(), square()); // Square
    result = std::pow(thrust::reduce(x2.begin(), x2.end()),0.5); // Sqrt of Sum
    return result;
}

double magnitude2(thrust::device_vector<double> x) {
    double result;
    result = std::sqrt(thrust::transform_reduce(x.begin(), x.end(), unary_square<double>(), 0.0, thrust::plus<double>())); // Square
    return result;
}

int main(int argc, char** argv) {
    if (argc != 2) {
        std::cerr << argv[0] << ": numero invalido de argumentos\n"; 
        std::cerr << "uso: " << argv[0] << "  tamanho do vetor\n"; 
        return 1;
    }
    int n = std::atoi(argv[1]); //numero de elementos
    steady_clock::time_point ts, te;

    // Faça um  vector em thrust 
    thrust::device_vector<double> x(n);
    // inicilize o  vector
    ts = steady_clock::now();
    std::generate(x.begin(),x.end(), std::rand);
    te = steady_clock::now();
    reportTime("Inicializacao", te - ts);
    // Calcule a magnitude do vetor
    ts = steady_clock::now();
    float len = magnitude(x);
    te = steady_clock::now();
    reportTime("Tempo para calculo", te - ts);

    std::cout << std::fixed << std::setprecision(4);
    std::cout << "Magnitude : " << len << std::endl;
    
    // Init
    // ts = steady_clock::now();
    // std::generate(x.begin(),x.end(), std::rand);
    // te = steady_clock::now();
    // reportTime("Inicializacao", te - ts);
    // Transform reduce
    ts = steady_clock::now();
    len = magnitude2(x);
    te = steady_clock::now();
    reportTime("Tempo para calculo", te - ts);


    std::cout << std::fixed << std::setprecision(4);
    std::cout << "Magnitude : " << len << std::endl;
}
